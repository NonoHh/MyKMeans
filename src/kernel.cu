#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void getDistance(double *cudaPoints, double *cudaClusters, int numPoints, int numClusters, int numThreadsInBlock, int numDims, double *pointsDistance)
{
    double result = 0;
    int blockID = blockIdx.x;
    if ((blockID + 1 == gridDim.x) && (numPoints % blockDim.x <= threadIdx.x)) {
        return;
    }
    for (int dIndex = 0; dIndex < numDims; dIndex++) {
        result += (cudaPoints[(blockID * numThreadsInBlock + threadIdx.x) * numDims + dIndex] - cudaClusters[threadIdx.y * numDims + dIndex])
            * (cudaPoints[(blockID * numThreadsInBlock + threadIdx.x) * numDims + dIndex] - cudaClusters[threadIdx.y * numDims + dIndex]);
    }
    // update distance
    pointsDistance[numPoints * threadIdx.y + (blockID * numThreadsInBlock + threadIdx.x)] = result;
}

__global__ void getMinDistance(int numPoints, int numClusters, int numThreadsInBlock, double *pointsDistance, int   *pointsBelong)
{
    int blockId = blockIdx.x;
    double minIndex = 0;
    if ((blockIdx.x == gridDim.x - 1) && (numPoints % blockDim.x <= threadIdx.x)) {
        return;
    }
    double minDistance = pointsDistance[(numThreadsInBlock * blockId) + threadIdx.x];
    for (int cIndex = 1; cIndex < numClusters; cIndex++) {
        double tmpDistance = pointsDistance[(numThreadsInBlock * blockId) + threadIdx.x + (cIndex * numPoints)];
        if (minDistance > tmpDistance) {
            minDistance = tmpDistance;
            minIndex = cIndex;
        }
    }
    // update belong
    pointsBelong[numThreadsInBlock * blockId + threadIdx.x] = minIndex;
}

__global__ void movePoints(double *cudaPoints, double *cudaVelocity, int numPoints, int numDims, int numThreadsInBlock, double dt)
{
    int blockID = blockIdx.x;
    if ((blockID + 1 == gridDim.x) && (numPoints % blockDim.x <= threadIdx.x)) {
        return;
    }
    for (int dIndex = 0; dIndex < numDims; dIndex++) {
        cudaPoints[(blockID * numThreadsInBlock + threadIdx.x) * numDims + dIndex] += dt * cudaVelocity[(blockID * numThreadsInBlock + threadIdx.x) * numDims + dIndex];
    }
}

hipError_t cudaMovePoints(double **points, double *cudaPoints, double *cudaVelocity, int numPoints, int numDims, double dt)
{
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int numThreadsInBlock = devProp.maxThreadsPerBlock;
    int numBlocks = numPoints / numThreadsInBlock;
    if (numPoints % numThreadsInBlock > 0) {
        numBlocks++;
    }
    movePoints << <numBlocks, numThreadsInBlock >> > (cudaPoints, cudaVelocity, numPoints, numDims, numThreadsInBlock, dt);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return cudaStatus;
    }
    cudaStatus = hipMemcpy((void**)points[0], cudaPoints, numPoints * numDims * sizeof(double), hipMemcpyDeviceToHost);
    return cudaStatus;
}

hipError_t setPointsBelong(double *cudaPoints, double **clusters, int numPoints, int numClusters, int numDims, int *pointsBelong)
{
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int numThreadsInBlock = devProp.maxThreadsPerBlock / numClusters;
    dim3 dim(numThreadsInBlock, numClusters);
    int numBlocks = numPoints / numThreadsInBlock;
    if (numPoints % numThreadsInBlock > 0) {
        numBlocks++;
    }
    double *cudaClusters;
    double *pointsDistance = 0;
    int *tmpPointsBelong = 0;
    hipError_t cudaStatus = hipMalloc((void**)&cudaClusters, numClusters * numDims * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed");
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&pointsDistance, numClusters * numPoints * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed");
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&tmpPointsBelong, numPoints * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed");
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    cudaStatus = hipMemcpy(cudaClusters, clusters[0], numClusters * numDims * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed");
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    getDistance << <numBlocks, dim >> > (cudaPoints, cudaClusters, numPoints, numClusters, numThreadsInBlock, numDims, pointsDistance);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize: error code %d\n", cudaStatus);
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    numThreadsInBlock = devProp.maxThreadsPerBlock;
    numBlocks = numPoints / numThreadsInBlock;
    if (numPoints % numThreadsInBlock > 0) { numBlocks++; }
    getMinDistance << <numBlocks, numThreadsInBlock >> > (numPoints, numClusters, numThreadsInBlock, pointsDistance, tmpPointsBelong);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel failed: %s\n", hipGetErrorString(cudaStatus));
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize: error code %d\n", cudaStatus);
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    cudaStatus = hipMemcpy(pointsBelong, tmpPointsBelong, numPoints * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed");
        cudaFreeAll(cudaClusters, pointsDistance, tmpPointsBelong);
        return cudaStatus;
    }
    return cudaStatus;
}

void cudaFreeAll(double *cudaClusters, double *pointsDistance, int *tmpPointsBelong)
{
    hipFree(cudaClusters);
    hipFree(pointsDistance);
    hipFree(tmpPointsBelong);
}
